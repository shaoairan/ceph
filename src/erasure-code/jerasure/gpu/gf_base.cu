#include "hip/hip_runtime.h"
#include "gf_base.h"

extern "C" {
#include "gf_bridge.h"
}


int copy_log_to_gpu_w8( /* void* log, void* anti_log, void* inv,*/ hipStream_t stream )
{
  char* log;
  char* anti_log;
  char* inv;

  get_w8_log_tables( log, anti_log, inv );
 
	hipMemcpyToSymbolAsync(HIP_SYMBOL(c_log), 		log,		GF_FIELD_SIZE_8_GPU, 	0, hipMemcpyHostToDevice, stream);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(c_antilog), 	anti_log,	GF_FIELD_SIZE_8_GPU*2, 	0, hipMemcpyHostToDevice, stream);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(c_inv), 		inv,		GF_FIELD_SIZE_8_GPU, 	0, hipMemcpyHostToDevice, stream);

	return 0;
}
