#include "hip/hip_runtime.h"
#include "clmsr_gpu.h"

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include "assert.h"
#include "jerasure_base.h"
#include "gf_base.h"
#include "math.h"

extern "C" {
#include "jerasure.h"
}

/*
 ============================================================================
 Name        : cudaTest.cu
 Author      : houyx
 Version     : 0.1
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#define talloc(type, num) (type *) malloc(sizeof(type)*(num))
#define A1A2_B1 1
#define A1B2_B1 2
#define B1A2_A1 3
#define A1B1_A2 4
#define B1B2_A1 5
#define GAMMA 6
#define GAMMA_INVERSE 7

#define debughouTab(num, fmt, arg...)     for(int o = 0; o < num; o ++) printf("\t");     printf((const char*)fmt, ##arg)


using namespace std;

//#define assert( a ) if((a) == false ) cout << "assert" <<  #a << " failed!\n", return -1



#define FT(A) FunctionTest4 printFunctionName(#A)

class FunctionTest4
{
  static int tabs;
  std::string a;
  public:
    FunctionTest4( std::string a_ ):a(a_)
    {

      for( int i = 0; i < tabs; i ++ )
      {
          printf("\t");
      }
      std::cout << "entering:: " << a << "\n";
      tabs ++;
    }

    ~FunctionTest4()
    {
      tabs --;
      for( int i = 0; i < tabs; i ++ )
      {
          printf("\t");
      }
      std::cout << "leave:: " << a << "\n";
    }
};

int FunctionTest4::tabs = 4;


static bool CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

bool ClmsrGpu::statusMark = true;


/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < vectorSize)
        data[idx] = 1.0/data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size)
{
    float *rc = new float[size];
    float *gpuData;

    CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
    CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));

    static const int BLOCK_SIZE = 256;
    const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
    reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

    CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(gpuData));
    return rc;
}

float *cpuReciprocal(float *data, unsigned size)
{
    float *rc = new float[size];
    for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
    return rc;
}


void initialize(float *data, unsigned size)
{
    for (unsigned i = 0; i < size; ++i)
        data[i] = 1.5*(i+1);
}

int docal()
{
    std::cout << "Glad to see I'm here in cuda.so->clmsr_gpu.cu \n";
    static const int WORK_SIZE = 65530;
    float *data = new float[WORK_SIZE];

    initialize (data, WORK_SIZE);

    float *recCpu = cpuReciprocal(data, WORK_SIZE);
    float *recGpu = gpuReciprocal(data, WORK_SIZE);
    float cpuSum = std::accumulate (recCpu, recCpu+WORK_SIZE, 0.0);
    float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);

    /* Verify the results */
    std::cout<<"gpuSum = "<<gpuSum<< " cpuSum = " <<cpuSum<<std::endl;

    /* Free memory */
    delete[] data;
    delete[] recCpu;
    delete[] recGpu;

    return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static bool CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
    if (err == hipSuccess)
        return true;
    else
    {
        std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
        ClmsrGpu::statusMark = false;
        exit(1);
    }
}

ClmsrProfile::ClmsrProfile( int q_,int t_,int d_,int sub_chunk_no_,\
        int k_,int m_,int w_,int nu_,\
        int gamma_,int* matrix_, \
        unsigned chunkSize_, unsigned subChunkSize_, MdsType mdsType_ ):\
        q(q_),t(t_),d(d_),\
        k(k_),m(m_),w(w_),nu(nu_),\
        gamma(gamma_),matrix(matrix_),\
        chunkSize(chunkSize_), subChunkSize(subChunkSize_), mdsType(mdsType_),sub_chunk_no(sub_chunk_no_)
        {
        }



ClmsrGpu::ClmsrGpu(  ClmsrProfile clmsrProfile_ ) : \
     clmsrProfile(clmsrProfile_)
     {
        printf("happy should I?");

        //done: init B_Buf and pin it
        CUDA_CHECK_RETURN(hipHostAlloc(&B_buf, clmsrProfile.q*clmsrProfile.t*sizeof(char*), hipHostMallocPortable));

        assert(B_buf != NULL);

        for(int i = 0; i < clmsrProfile.q*clmsrProfile.t; i++)
        {
            //checkCudaErrors(hipHostAlloc(&h_data_in[i], memsize, hipHostMallocPortable));
            if(B_buf[i]==NULL)
            {
                CUDA_CHECK_RETURN(hipHostAlloc(&B_buf[i], (size_t)(clmsrProfile.subChunkSize*clmsrProfile.sub_chunk_no), hipHostMallocPortable));
                assert(B_buf[i]!= NULL);
            }
        }

        //done: get GpuInfo

        //todo try to put it in the shared memory;
        CUDA_CHECK_RETURN(hipMalloc(&matrix_gpu, sizeof(int)*clmsrProfile.k*clmsrProfile.m));
        CUDA_CHECK_RETURN(hipMemcpy(matrix_gpu, clmsrProfile.matrix, sizeof(int)*(clmsrProfile.k*clmsrProfile.m), hipMemcpyHostToDevice));

     }

int ClmsrGpu::pinAllMemoryForRepair(  map<int,char*>& repaired_data, int sizeRepair,  map<int,char*>& helper_data,  int sizeHelper )
{
    pinMemory(repaired_data, sizeRepair );
    pinMemory(helper_data, sizeHelper );
    return 0;
}

int ClmsrGpu::unpinAllMemoryForRepair(  map<int,char*>& repaired_data,   map<int,char*>& helper_data )
{
    unpinMemory(repaired_data );
    unpinMemory(helper_data );
    return 0;

}

int ClmsrGpu::pinAllMemoryForDecode(  char** data_ptrs, int sizeData,  char** code_ptrs,  int sizeCode )
{   

    //todo: check nu is alloced? may the author is wrong and forget to do it, error!!!
    for( int i = 0; i < clmsrProfile.k + clmsrProfile.nu; i ++ )
    {       //todo:flags
        CUDA_CHECK_RETURN(hipHostRegister(data_ptrs[i], sizeData, hipHostMallocPortable));
    }

    for( int i = 0; i < clmsrProfile.m; i ++ )
    {       //todo:flags
        CUDA_CHECK_RETURN(hipHostRegister(code_ptrs[i], sizeCode, hipHostMallocPortable));
    }
    return 0;

}

int ClmsrGpu::unpinAllMemoryForDecode(  char** data_ptrs,   char** code_ptrs )
{
    for( int i = 0; i < clmsrProfile.k + clmsrProfile.nu; i ++ )
    {       //todo:flags
        CUDA_CHECK_RETURN(hipHostUnregister(data_ptrs[i]));
    }

    for( int i = 0; i < clmsrProfile.m; i ++ )
    {       //todo:flags
        CUDA_CHECK_RETURN(hipHostUnregister(code_ptrs[i]));
    }
    return 0;
}

ClmsrGpu::~ClmsrGpu()
{
    //todo: free B_buf and unpin it
    for(int i = 0; i < clmsrProfile.q*clmsrProfile.t; i++)
    {
        //checkCudaErrors(hipHostAlloc(&h_data_in[i], memsize, hipHostMallocPortable));
        hipHostFree(B_buf[i]);
    }

    hipHostFree(B_buf);

    hipFree(matrix_gpu);
}


inline void ClmsrGpu::pinMemory( map<int,char*> map, int size )
{
    for( std::map<int,char*>::iterator iter = map.begin(); iter != map.end(); iter++) {
        //todo:flags
        CUDA_CHECK_RETURN(hipHostRegister(iter->second, size, hipHostMallocPortable));
    }
}

inline void ClmsrGpu::unpinMemory( map<int,char*> map )
{
    for(std::map<int,char*>::iterator iter = map.begin(); iter != map.end(); iter++) {
        CUDA_CHECK_RETURN(hipHostUnregister(iter->second));
    }
}

DeviceInfo::DeviceInfo()
{
    CUDA_CHECK_RETURN(hipGetDeviceCount(&deviceCount));
    device = new hipDeviceProp_t[deviceCount];
    for( int i = 0; i < deviceCount; i ++ )
    {
        CUDA_CHECK_RETURN(hipSetDevice(i));
        CUDA_CHECK_RETURN(hipGetDeviceProperties(&(device[i]),i));
    }
}

DeviceInfo::~DeviceInfo()
{
    free(device);
}

SingleGpuRoute::SingleGpuRoute( int deviceId_, ClmsrGpu* ClmsrGpuP_, int subSubChunkStart_, int subSubChunkSize_ ): \
clmsrGpuP(ClmsrGpuP_), deviceId(deviceId_),deviceProp(((ClmsrGpuP_->deviceInfo).device[deviceId_])),\
subSubChunkStart(subSubChunkStart_), subSubChunkSize(subSubChunkSize_), clmsrProfileP(&(ClmsrGpuP_->clmsrProfile))\
{
    CUDA_CHECK_RETURN(hipSetDevice(deviceId));


    pieceKernelGridSize = deviceProp.multiProcessorCount;
    pieceKernelBlockSize = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);

    planeKernelGridSize = deviceProp.multiProcessorCount;
    planeKernelBlockSize = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);


    int layerSize = subSubChunkSize * clmsrProfileP->q * clmsrProfileP->t;
    //todo: check if overflowed: now because layer 1's A1 is need, B1 is caled, couple A2 is need, B2 is caled, so at most is 4 times memory.
    if(float(deviceProp.totalGlobalMem) < float(layerSize) * 4.0 + clmsrProfileP->k*clmsrProfileP->m*sizeof(int) )
    {
        pieceCount = layerSize*4/deviceProp.totalGlobalMem + 1;
    }
    else
    {
        pieceCount = 1;
    }

    init();

    CUDA_CHECK_RETURN( hipMalloc(& gf_table.g_log, GF_FIELD_SIZE_8_GPU) );
    CUDA_CHECK_RETURN( hipMalloc(& gf_table.g_anti_log, GF_FIELD_SIZE_8_GPU*2) );
    CUDA_CHECK_RETURN( hipMalloc(& gf_table.g_inv, GF_FIELD_SIZE_8_GPU) );

}

SingleGpuRoute::~SingleGpuRoute()
{
    deinit();
}


int SingleGpuRoute::__getPieceSize( int i )
{
    //error!!!!
    //todo: think about thread
    int baseSize = subSubChunkSize/pieceCount;
    //int baseSize = subSubChunkSize%pieceCount == 0;


    if( subSubChunkSize%pieceCount == 0 )
    {
        return baseSize;
    }
    else if( (i + 1) <= subSubChunkSize%pieceCount )
    {
        baseSize ++;
    }

    return baseSize;
}

void SingleGpuRoute::init()
{
    FT(SingleGpuRoute::init);

    for( int i = 0; i < STREAM_NUM; i ++ )
    {
        CUDA_CHECK_RETURN(hipStreamCreate(&streams[i]));
    }

    for( int i = 0; i < EVENT_NUM; i ++ )
    {
        CUDA_CHECK_RETURN(hipEventCreate(&events[i]));
    }
    //hipEventRecord(events[i], streams[i]);
    //todo: init stream and events
}


inline void get_plane_vector(int q, int t, int z, int* z_vec)
{
  int i ;

  for(i = 0; i<t; i++ ){
    z_vec[t-1-i] = z%q;
    z = (z - z_vec[t-1-i])/q;
  }
  return;
}

__global__ void testGf( gf_w8_log_gpu gf_table, int w )
{
    unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned threadsNum = gridDim.x*blockDim.x;

    set_gf_table(gf_table);

    if( idx == 0 )
    {
        printf("=========In testGf=============mul\n");
        for( int i = 0; i < 256; i ++ )
        {
            for( int j = 0; j < 256; j ++ )
            {
                printf( "%d\t*\t%d\t=\t%d\n", i, j, galois_single_multiply_gpu_logtable_w8((unsigned char)i, (unsigned char)j) );
            }
        }
        printf("=========In testGf=============div\n");
        for( int i = 0; i < 256; i ++ )
        {
            for( int j = 1; j < 256; j++ )
            {
                printf( "%d\t/\t%d\t=\t%d\n", i, j, galois_single_divide_gpu_logtable_w8((unsigned char)i, (unsigned char)j) );
            }
        }
    }
}

__global__ void pieceKernelGamma( gf_w8_log_gpu gf_table, unsigned char gamma, unsigned char** dataPt,  int nodeId, int calType, int dataSize, int patchSize, int w )
{
    set_gf_table(gf_table);

    unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned threadsNum = gridDim.x*blockDim.x;

    unsigned char tmatrix[4];

    if( idx == 0 )
    {
        debughouTab(3,"\n\n=============\ndataSize: %d\tpatchSize: %d\tw: %d\tnodeId: %d\t\n===========\n\n", dataSize,patchSize,w,nodeId);
    }


    unsigned char* A[2];
    unsigned char* dest[2];

    switch(calType)
    {
        case GAMMA :

            tmatrix[0] = 1;
            tmatrix[1] = gamma;
            tmatrix[2] = gamma;
            tmatrix[3] = 1;

            A[0] = dataPt[nodeId] + patchSize*0; //B2
            A[1] = dataPt[nodeId] + patchSize*1; //B1

            dest[0] = dataPt[nodeId] + patchSize*2;
            dest[1] = dataPt[nodeId] + patchSize*3;

            break;

        case GAMMA_INVERSE  :

            //char gamma_square = galois_single_multiply_gpu_logtable_w8(gamma, gamma);
            //char gamma_det_inv = galois_single_divide_gpu_logtable_w8(1, 1 ^ (gamma_square));
            //tmatrix[0] = gamma_det_inv;      
            //tmatrix[1] = galois_single_multiply_gpu_logtable_w8(gamma,gamma_det_inv);      
            //char gamma_square = galois_single_multiply_gpu_logtable_w8(gamma, gamma);
            //char gamma_det_inv = galois_single_divide_gpu_logtable_w8(1, 1 ^ (galois_single_multiply_gpu_logtable_w8(gamma, gamma)));
            tmatrix[0] = galois_single_divide_gpu_logtable_w8(1, 1 ^ (galois_single_multiply_gpu_logtable_w8(gamma, gamma)));
            tmatrix[1] = galois_single_multiply_gpu_logtable_w8(gamma,galois_single_divide_gpu_logtable_w8(1, 1 ^ (galois_single_multiply_gpu_logtable_w8(gamma, gamma))));
            tmatrix[2] = tmatrix[1];
            tmatrix[3] = tmatrix[0];



            A[0] = dataPt[nodeId] + patchSize*2; //B2
            A[1] = dataPt[nodeId] + patchSize*3; //B1

            dest[0] = dataPt[nodeId] + patchSize*0;
            dest[1] = dataPt[nodeId] + patchSize*1;

            break;

        default:
            printf("error Case %d\n", calType);
            __threadfence_system();
            asm("trap;");
            break;
    }


    //jerasure_matrix_dotprod(2, w, &tmatrix[0], NULL, 2, A, dest, size);
    //jerasure_matrix_dotprod(2, w, &tmatrix[2], NULL, 3, A, dest, size);

    for( int i = idx; i < dataSize; i += threadsNum )
    {
        //mark = 0;
        //todo: optmize a*1 case;
        dest[0][i] = galois_single_multiply_gpu_logtable_w8(A[0][i], tmatrix[0]) ^ galois_single_multiply_gpu_logtable_w8(A[1][i], tmatrix[1]);
        dest[1][i] = galois_single_multiply_gpu_logtable_w8(A[0][i], tmatrix[2]) ^ galois_single_multiply_gpu_logtable_w8(A[1][i], tmatrix[3]);
    }
}

__global__ void pieceKernel( gf_w8_log_gpu gf_table, unsigned char gamma, unsigned char** dataPt,  int nodeId, int calType, int dataSize, int patchSize, int w )
{
    set_gf_table(gf_table);

    unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned threadsNum = gridDim.x*blockDim.x;

    if( idx == 0 )
    {
        debughouTab(3,"\n\n=============\ndataSize: %d\tpatchSize: %d\tw: %d\tnodeId: %d\tthreadsNum: %d\n===========\n\n", dataSize,patchSize,w,nodeId,threadsNum);
    }


    unsigned char tmatrix[2];
    unsigned char* in_dot[2];
    unsigned char* dest;
    //todo: make the preceed all the same by changing the pointer and matrix;
    //for( int i = idx; i < dataSize; i += threadsNum )
    //todo: if use this patern, you must make sure that blockDim.x = blockSize;
/*    for( int i = blockStart + threadIdx.x; i < dataSize && i < blockStart + dataSizeBlock; i += blockDim.x )
    {

    }*/

    switch(calType)
    {
        case A1A2_B1        :
            if( idx == 0 )
            {
                debughouTab(1, "in A1A2_B1: %d\n", dataSize);
            }

            tmatrix[0] = 1;
            tmatrix[1] = gamma;

            in_dot[0] = dataPt[nodeId];  //A1
            in_dot[1] = dataPt[nodeId] + patchSize; //A2

            dest = dataPt[nodeId] + patchSize*2; //B1
            //debughouTab(1, "case finished %d\n", idx);
            break;

        case A1B2_B1        :

            if( idx == 0 )
            {
                debughouTab(1, "in A1B2_B1: %d\n", dataSize);
            }

            //char gamma_square = galois_single_multiply_gpu_logtable_w8(gamma, gamma);

            //tmatrix[0] = (1 ^ gamma_square);
            tmatrix[0] = (1 ^ galois_single_multiply_gpu_logtable_w8(gamma, gamma));
            tmatrix[1] = gamma;

            in_dot[0] = dataPt[nodeId];  //A1
            in_dot[1] = dataPt[nodeId] + patchSize*3; //B2

            dest = dataPt[nodeId] + patchSize*2; //B1
            //debughouTab(1, "case finished %d\n", idx);
            break;

        case B1A2_A1        :

            if( idx == 0 )
            {
                debughouTab(1, "in B1A2_A1: %d\n", dataSize);
            }

            tmatrix[0] = 1;
            tmatrix[1] = gamma;

            in_dot[0] = dataPt[nodeId] + patchSize*2;  //B1
            in_dot[1] = dataPt[nodeId] + patchSize*1; //A2

            dest = dataPt[nodeId]; //A1

            //debughouTab(1, "case finished %d\n", idx);
            break;

        case A1B1_A2        :

            if( idx == 0 )
            {
                debughouTab(1, "in A1B1_A2: %d\n", dataSize);
            }


            tmatrix[0] = galois_single_divide_gpu_logtable_w8(1,gamma);
            tmatrix[1] = tmatrix[0];

            in_dot[0] = dataPt[nodeId] + patchSize*2;  //B1
            in_dot[1] = dataPt[nodeId] + patchSize*1; //A2

            dest = dataPt[nodeId]; //A1

            //debughouTab(1, "case finished %d\n", idx);
            break;       


        case B1B2_A1        :

            if( idx == 0 )
            {
                debughouTab(1, "in A1B1_A2: %d\n", dataSize);
            }


            tmatrix[0] = galois_single_divide_gpu_logtable_w8(1, 1 ^ (galois_single_multiply_gpu_logtable_w8(gamma, gamma)));
            tmatrix[1] = galois_single_multiply_gpu_logtable_w8(gamma,galois_single_divide_gpu_logtable_w8(1, 1 ^ (galois_single_multiply_gpu_logtable_w8(gamma, gamma))));
            

            in_dot[0] = dataPt[nodeId] + patchSize*2;  //B1
            in_dot[1] = dataPt[nodeId] + patchSize*3; //B2

            dest = dataPt[nodeId]; //A1
            //debughouTab(1, "case finished %d\n", idx);
            break;

        default:
            
            printf("error Case %d\n", calType);
            __threadfence_system();
            asm("trap;");
            break;
    }
    
/*    __threadfence_system();
    __syncthreads();
    debughouTab(1, "ready for loooooop %d\n", idx);
    __syncthreads();
    __threadfence_system();*/
/*
    if( idx == 0 )
    {
        for( int i = 0; i < 256; i ++ )
        {
            debughouTab(1, "&====: %d: %d----%d\n", i, sh_log[i], sh_antilog[i]);
        }

        for( int i = 256; i < 256*2; i ++ )
        {
            debughouTab(1, "&==&==: %d: %d\n", i, sh_log[i] );            
        }
    }*/


    //print_table( idx, sh_log, sh_antilog, sh_inv );
    //jerasure_matrix_dotprod(2, w, &tmatrix[0], NULL, 2, in_dot, dest, size);
    //dest[i] = in_dot[0][i] * tmatrix[0] + in_dot[1][i] * tmatrix[1]; 
    for( int i = idx; i < dataSize; i += threadsNum )
    {
        /*debughouTab(1,"&&& findme!! idx: %d\ti: %d\n", idx, i);
        debughouTab(1,"&&& findme!! in_dot[0][i]: %u\n", (unsigned char) in_dot[0][i]);
        debughouTab(1,"&&& findme!! tmatrix[0]: %u\n", (unsigned char) tmatrix[0]);
        debughouTab(1,"&&& findme!! dest[i]: %u\n", (unsigned char) dest[i]);
        debughouTab(1,"&&& findme!! in_dot[1][i]: %u\n", (unsigned char) in_dot[0][i]);
        debughouTab(1,"&&& findme!! tmatrix[1]: %u\n", (unsigned char) tmatrix[1]);
        debughouTab(1,"&&& findme!! sh_log[in_dot[0][i]]: %u\n", (unsigned char) sh_log[in_dot[0][i]]);
        debughouTab(1,"&&& findme!! sh_log[in_dot[1][i]]: %u\n", (unsigned char) sh_log[in_dot[1][i]]);

        debughouTab(1,"&&& findme!! sh_log[1]: %u\n", (unsigned char) sh_log[1]);
        debughouTab(1,"&&& findme!! sh_log[2]: %u\n", (unsigned char) sh_log[2]);

        debughouTab(1,"&&& findme!! sh_log[tmatrix[0]]: %u\n", (unsigned char) sh_log[(unsigned char) tmatrix[0]]);
        debughouTab(1,"&&& findme!! sh_log[tmatrix[1]]: %u\n", (unsigned char) sh_log[(unsigned char) tmatrix[1]]);
        debughouTab(1,"&&& findme!! sh_log[tmatrix[0]]: %u\n", (unsigned char) sh_log[tmatrix[0]]);
        debughouTab(1,"&&& findme!! sh_log[tmatrix[1]]: %u\n", (unsigned char) sh_log[tmatrix[1]]);
        debughouTab(1,"&&& findme!! w8(in_dot[0][i], tmatrix[0]): %u\n", galois_single_multiply_gpu_logtable_w8(in_dot[0][i], tmatrix[0]));
        debughouTab(1,"&&& findme!! w8(in_dot[1][i], tmatrix[1]): %u\n", galois_single_multiply_gpu_logtable_w8(in_dot[1][i], tmatrix[1]));
        */
        //todo: optmize a*1 case;
        dest[i] = galois_single_multiply_gpu_logtable_w8(in_dot[0][i], tmatrix[0]) ^ galois_single_multiply_gpu_logtable_w8(in_dot[1][i], tmatrix[1]);    
    }
}



/*planeKernel<<<planeKernelGridSize,planeKernelBlockSize,streams[1]>>>(\
                        clmsrProfileP->k, clmsrProfileP->nu, clmsrProfileP->m, clmsrProfileP->w,\
                        clmsrGpuP->matrix_gpu, erasure_locations_gpu, planePOnGpuK, pieceSize
                     );*/

__global__ void planeKernel(gf_w8_log_gpu gf_table,  const int k, int nu, const int m, int w, int q, int t, int* matrix, int* decode_matrix, \
    int* dm_ids, int* erasure_loc_data, int erased_data_size, int* erasure_loc_coding, int erased_coding_size, unsigned char** dataPt, int dataSize, int patchSize )
{
    set_gf_table( gf_table );

    unsigned idx = blockIdx.x*blockDim.x+threadIdx.x;
    unsigned threadsNum = gridDim.x*blockDim.x;

    //load matrix to gpu shared memery;
    extern __shared__ unsigned char s[];
    unsigned char *sh_decode_matrix = s;                        //
    unsigned char *sh_matrix = (unsigned char*)&sh_decode_matrix[k*k]; // 

    for( int t = idx; t < k*k; t += threadsNum )
    {
        sh_decode_matrix[t] = (unsigned char) decode_matrix[t];
    }


    //todo: optmize and merge
    for( int t = idx; t < k*m; t += threadsNum )
    {
        sh_matrix[t] = (unsigned char) matrix[t];
    }
    
    __syncthreads();

/*  for (i = 0; i < k; i++) {
    if (erased[i]) {
      jerasure_matrix_dotprod_substripe(k, w, decoding_matrix+(i*k), dm_ids, i, data_ptrs, coding_ptrs, z, ss_size);
      edd--;
    }
  }

   Finally, re-encode any erased coding devices 

  for (i = 0; i < m; i++) {
    if (erased[k+i]) {
      jerasure_matrix_dotprod_substripe(k, w, matrix+(i*k), NULL, i+k, data_ptrs, coding_ptrs, z, ss_size);
    }
  }*/

    unsigned char * data_now;
    unsigned char * erasure_now;

    for( int i = 0 ; i < k; i ++ )
    {
        data_now = dataPt[dm_ids[i]] + patchSize*2;//B1 is what we want do RS

        for( int j = 0; j < erased_data_size; j ++ )
        {
            erasure_now = dataPt[erasure_loc_data[j]] + patchSize*2;
            
            for( int t = idx; t < dataSize; t += threadsNum )
            {
                //todo: ensure that dest[i] is all clean first time;
                if( i == 0 )
                {
                    erasure_now[t] = galois_single_multiply_gpu_logtable_w8( data_now[t], (unsigned char)(sh_decode_matrix + j*k + i));
                }
                else
                {
                    erasure_now[t] = galois_single_multiply_gpu_logtable_w8( data_now[t], (unsigned char)(sh_decode_matrix + j*k + i));
                }    
            }
        }
    }

    for( int i = 0 ; i < k; i ++ )
    {
        data_now = dataPt[i] + patchSize*2;//B1 is what we want do RS

        for( int j = 0; j < erased_coding_size; j ++ )
        {
            erasure_now = dataPt[erasure_loc_coding[j]] + patchSize*2;
            
            for( int t = idx; t < dataSize; t += threadsNum )
            {
                //todo: ensure that dest[i] is all clean first time;
                if( i == 0 )
                {
                    erasure_now[t] = galois_single_multiply_gpu_logtable_w8( data_now[t], (unsigned char)(sh_matrix + j*k + i));
                }
                else
                {
                    erasure_now[t] = galois_single_multiply_gpu_logtable_w8( data_now[t], (unsigned char)(sh_matrix + j*k + i));
                }    
            }
        }
    }
}



int SingleGpuRoute::doRepair( map<int,char*> &repaired_data, set<int> &aloof_nodes,
                           map<int, char*> &helper_data, int repair_blocksize, map<int,int> &repair_sub_chunks_ind, char** B_buf )
{
    FT(SingleGpuRoute::doRepair);

    assert( 1 == 0 );
    cout << "\n\n\nsurprise~\n\n\n" << endl;


    init_gf_log_w8_gpu();

    debughouTab(1, "repaired_data:**************\n");
    for( map<int,char*>::iterator it = repaired_data.begin(); it != repaired_data.end(); it ++ )
    {
        debughouTab(1, "%d,", it->first );
    }
    debughouTab(1, "\n**************\n");

    debughouTab(1, "helper_data:**************\n");
    for( map<int,char*>::iterator it = helper_data.begin(); it != helper_data.end(); it ++ )
    {
        debughouTab(1, "%d,", it->first);
    }
    debughouTab(1, "\n**************\n");

    debughouTab(1, "aloof_nodes:**************\n");
    for( set<int>::iterator it = aloof_nodes.begin(); it != aloof_nodes.end(); it ++ )
    {
        debughouTab(1, "%d,", *it);
    }
    debughouTab(1, "\n**************\n");


    debughouTab(1, "repair_sub_chunks_ind:**************\n");
    for( map<int,int>::iterator it = repair_sub_chunks_ind.begin(); it != repair_sub_chunks_ind.end(); it ++ )
    {
        debughouTab(1, "%d: %d;\t", it->first, it->second);
    }
    debughouTab(1, "\n**************\n");

    printf(" k: %d\t nu: %d\t m: %d\n",clmsrProfileP->k, clmsrProfileP->nu, clmsrProfileP->m );
    const int k = clmsrProfileP->k + clmsrProfileP->nu;
    const int m = clmsrProfileP->m;

    const int sub_chunksize = clmsrProfileP->subChunkSize;
    const int q = clmsrProfileP->q, t = clmsrProfileP->t;
    const int qt = q * t;
    int* z_vec;

    CUDA_CHECK_RETURN( hipHostAlloc(&z_vec, t * sizeof(int), hipHostMallocPortable) );

    map<int, set<int> > ordered_planes;
    map<int, int> repair_plane_to_ind;
    int order = 0;
    int x,y, node_xy, node_sw, z_sw;
    char *A1, *A2, *B1, *B2;
    int count_retrieved_sub_chunks = 0;
    int num_erased = 0;

    int *decode_matrix_gpu;
    int *decode_matrix;
    int *dm_ids;
    int *dm_ids_gpu;
    int *erased;
    int *erasure_loc_data;
    int *erasure_loc_data_gpu;
    int *erasure_loc_coding;
    int *erasure_loc_coding_gpu;
    int erased_data_size = 0;
    int erased_coding_size = 0;
    
    bool init_matrix = false;

    dm_ids = talloc(int, k);
    decode_matrix = talloc(int, k*k);
    erased = talloc(int, k + m);
    erasure_loc_data = talloc(int, qt);
    erasure_loc_coding = talloc(int, qt);

    CUDA_CHECK_RETURN(hipMalloc(&decode_matrix_gpu, k * k *sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&dm_ids_gpu, k * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&erasure_loc_data_gpu, qt * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&erasure_loc_coding_gpu, qt * sizeof(int)));

    //malloc the data temp space on the gpu a piece has 4 things: A1,A2,B1,B2;
    
    int pieceSizeMax = __getPieceSize(0);
    unsigned char* planeOnGpu[qt];
    for( int i = 0; i < qt; i ++ )
    {
        CUDA_CHECK_RETURN(hipMalloc(&planeOnGpu[i], pieceSizeMax*4));
    }

    unsigned char** planePOnGpuK;

    CUDA_CHECK_RETURN(hipMalloc(&planePOnGpuK, qt*sizeof(unsigned char*)));

    CUDA_CHECK_RETURN(hipMemcpy(planePOnGpuK, planeOnGpu, qt*sizeof(unsigned char*), hipMemcpyHostToDevice));

    int plane_count = 0;
    int erasure_locations[qt];

    //get order of all planes
    for(map<int,int>::iterator i = repair_sub_chunks_ind.begin(); i != repair_sub_chunks_ind.end(); ++i)
    {
        get_plane_vector(q,  t, i->second, z_vec);
        order = 0;
        //check across all erasures
        for(map<int,char*>::iterator j = repaired_data.begin(); j != repaired_data.end(); ++j)
        {
          if(j->first% q == z_vec[j->first/q])order++;
        }
        assert(order>0);
        ordered_planes[order].insert(i->second);
        repair_plane_to_ind[i->second] = i->first;
    }

    debughouTab(1, "repair_plane_to_ind:**************\n");
    for( map<int,int>::iterator it = repair_plane_to_ind.begin(); it != repair_plane_to_ind.end(); it ++ )
    {
      debughouTab(1, "%d: %d;\t", it->first, it->second);
    }
    debughouTab(1, "\n**************\n");


    debughouTab(1, "ordered_planes:**************\n");
    for(order=1; ;order++){
        if(ordered_planes.find(order) == ordered_planes.end())
        {
            break;
        }
        else
        {
            debughouTab(0, "\norder: %d\n",order);
            for( set<int>::iterator it = ordered_planes[order].begin(); it != ordered_planes[order].end(); it ++ )
            {
                debughouTab(1, "%d, ", *it);
            }
            debughouTab(0, "\n");
        }
    }
    debughouTab(1, "\n**************\n");


    //repair planes in order
    for(order=1; ;order++){
        if(ordered_planes.find(order) == ordered_planes.end())
        {
            break;
        }
        else
        {
            plane_count += ordered_planes[order].size();
            debughouTab(0, "\n\n\n&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&\norder: %d\n&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&&\n\n\n",order);

            //where GPU works
            for(set<int>::iterator z=ordered_planes[order].begin(); z != ordered_planes[order].end(); ++z)
            {
                get_plane_vector(q,t,*z, z_vec);

                debughouTab(0, "\n\n**************************************\nplane now: %d\nz_vec\n", *z);

                for( int u = 0; u < t; u ++ )
                {
                    debughouTab(1, "%d", z_vec[u]);
                }
                debughouTab(0, "\n***************************************\n\n\n");

                int pieceOffset = 0;
                int pieceSize = 0;
                init_matrix = false;

                //printf("get set<int>::iterator z=ordered_planes[order].begin(); z != ordered_planes[order].end()=========================================\n");

                printf("pieceCount %d \n", pieceCount );

                for( int pi = 0; pi < pieceCount; pi ++ )
                {
                    pieceSize = __getPieceSize(pi);

                    debughouTab(0,"\n\n***********************\npieceCount: %d\tpieceSize: %d\tpieceSizeMax: %d\tpi: %d\tpieceoffset: %d\n***************************\n\n",\
                        pieceCount,pieceSize,pieceSizeMax,pi,pieceOffset);

                    num_erased = 0;
                    for(y=0; y < t; y++)
                    {
                        for(x = 0; x < q; x++)
                        {


                            node_xy = y*q + x;//todo: check pow is right
                            z_sw = (*z) + (x - z_vec[y])*(int)(pow(q,t-1-y));
                            node_sw = y*q + z_vec[y];

                            debughouTab(1, "node_xy: %d\t node_sw: %d\t z_sw: %d\t q: %d\tx: %d\t y: %d\n", node_xy,node_sw,z_sw,q,x,y);

                            if( (repaired_data.find(node_xy) != repaired_data.end()) )
                            {//case of erasure, aloof node can't get a B.

                                erasure_locations[num_erased] = node_xy;
                                num_erased++;

                                if( repaired_data.find(node_sw) != repaired_data.end() )
                                {//node_sw must be a helper
                                    if(x > z_vec[y])//todo: check if is right
                                    {
                                        B2 = &B_buf[node_sw][repair_plane_to_ind[z_sw]*sub_chunksize + pieceOffset];
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax*3,  B2, pieceSize, hipMemcpyHostToDevice) );
                                    }
                                }
                                else
                                {
                                    assert( helper_data.find(node_sw) != helper_data.end() );
                                    A2 = &helper_data[node_sw][repair_plane_to_ind[z_sw]*sub_chunksize + pieceOffset];

                                    if( z_vec[y] != x)
                                    {
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax*1, A2, pieceSize, hipMemcpyHostToDevice) );
                                        //get_B1_fromA1A2(&B_buf[node_xy][repair_plane_to_ind[*z]*sub_chunksize], A1, A2, sub_chunksize);
                                    }
                                }

                            }
                            else if( (aloof_nodes.find(node_xy) != aloof_nodes.end()) )
                            {
                                erasure_locations[num_erased] = node_xy;
                                num_erased++;
                            }
                            else
                            {//should be in helper data
                                assert(helper_data.find(node_xy) != helper_data.end());
                                //so A1 is available, need to check if A2 is available.
                                A1 = &helper_data[node_xy][repair_plane_to_ind[*z]*sub_chunksize + pieceOffset];

                                
                                //dout(10) << "current node=" << node_xy << " plane="<< *z << " node_sw=" << node_sw << " plane_sw="<< z_sw << dendl;
                                //consider this as an erasure, if A2 not found.
                                //todo: 这里的判断条件不太对啊， 没有判断你是一个helper 但你要的A2在erasred里的情况
                                if(repair_plane_to_ind.find(z_sw) == repair_plane_to_ind.end())
                                {
                                    debughouTab(3, "put node_xy: %d in erasures for z_sw: %d lost\n", node_xy, z_sw);
                                    erasure_locations[num_erased] = node_xy;
                                    //dout(10)<< num_erased<< "'th erasure of node " << node_xy << " = (" << x << "," << y << ")" << dendl;
                                    num_erased++;
                                }
                                else
                                {
                                    if(repaired_data.find(node_sw) != repaired_data.end())
                                    {
                                        //todo: 尝试在这里加assert证明repaired_data一定被算出来了，解决上面的todo
                                        assert(z_sw < sub_chunk_no);
                                        A2 = &repaired_data[node_sw][z_sw*sub_chunksize + pieceOffset];

                                        //transfer A1A2 and cal B1 B2;
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy],                A1, pieceSize, hipMemcpyHostToDevice) );
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax, A2, pieceSize, hipMemcpyHostToDevice) );
                                        
                                        //todo: find a parameter
                                        pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table,  clmsrProfileP->gamma,planePOnGpuK,node_xy, A1A2_B1, pieceSize, pieceSizeMax, clmsrProfileP->w);

                    //debug
                    CUDA_CHECK_RETURN( hipDeviceSynchronize() );

                                        //get_B1_fromA1A2(&B_buf[node_xy][repair_plane_to_ind[*z]*sub_chunksize], A1, A2,sub_chunksize);
                                    }
                                    else if(aloof_nodes.find(node_sw) != aloof_nodes.end())
                                    {
                                        B2 = &B_buf[node_sw][repair_plane_to_ind[z_sw]*sub_chunksize + pieceOffset];

                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy],                  A1, pieceSize, hipMemcpyHostToDevice) );
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax*3, B2, pieceSize, hipMemcpyHostToDevice) );
                                        
                                        //todo: find a parameter
                                        pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,node_xy,A1B2_B1, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                        
                    //debug
                    CUDA_CHECK_RETURN( hipDeviceSynchronize() );
                                        //get_B1_fromA1B2(&B_buf[node_xy][repair_plane_to_ind[*z]*sub_chunksize], A1, B2, sub_chunksize);
                                    }
                                    else
                                    {
                                        assert(helper_data.find(node_sw) != helper_data.end());
                      //dout(10) << "obtaining B1 from A1 A2 for node: " << node_xy << " on plane:" << *z << dendl;
                                        A2 = &helper_data[node_sw][repair_plane_to_ind[z_sw]*sub_chunksize + pieceOffset];
                                        if( z_vec[y] != x)
                                        {
                                            CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy],                  A1, pieceSize, hipMemcpyHostToDevice) );
                                            CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax*1, A2, pieceSize, hipMemcpyHostToDevice) );
                                        
                                            //todo: find a parameter
                                            pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,node_xy,A1A2_B1, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                            
                    //debug
                    CUDA_CHECK_RETURN( hipDeviceSynchronize() );
                                            //get_B1_fromA1A2(&B_buf[node_xy][repair_plane_to_ind[*z]*sub_chunksize], A1, A2, sub_chunksize);
                                        }
                                        else
                                        {
                                            //CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy],                  A1, pieceSize, hipMemcpyHostToDevice) );
                                            CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax*2, A1, pieceSize, hipMemcpyHostToDevice) );
                                            

                    //debug
                    CUDA_CHECK_RETURN( hipDeviceSynchronize() );
                                            //pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,streams[1]>>>(planePOnGpuK,node_xy,A1A2);
                                            //pieceKernel<<<>>>(A1A2);red point
                                            //memcpy(&B_buf[node_xy][repair_plane_to_ind[*z]*sub_chunksize], A1, sub_chunksize);
                                        }
                                    }
                                }

                            }


                        }//y
                    }//x

                    erasure_locations[num_erased] = -1;
                    //int erasuresxy[num_erased];
                    //get_erasure_coordinates(erasure_locations, erasuresxy, num_erased);
                    //we obtained all the needed B's
                    assert(num_erased <= m);
                    
                    int ppp;

                    debughouTab(4, "erasures_location out is: ==================================\n");
                    for (ppp = 0; erasure_locations[ppp] != -1; ppp++) {
                        debughouTab(4,"%d,", erasure_locations[ppp] );
                    }   
                    debughouTab(0,"\n");

                    debughouTab( 4 , "num_erased: %d\t m: %d\t, erasure_locations[num_erased]: %d\n", num_erased, m, erasure_locations[num_erased]  );

                    if(!init_matrix)
                    {
                        init_matrix = true;
                        
                        if(erasures_to_erased_gpu (k, m, erasure_locations, erased ) < 0 )
                        {
                            printf("haha, you get an error when calling erasures_to_erased_gpu!\n");
                            return -1;
                        }
                        //erased = jerasure_erasures_to_erased (k, m, erasure_locations );

                        printf("get before jerasure_make_decoding_matrix=========================================\n");
                        if (jerasure_make_decoding_matrix(k, m, clmsrProfileP->w, clmsrProfileP->matrix, erased, decode_matrix, dm_ids) < 0) 
                        {
                          printf("Can not get decoding matrix!!!\n");
                          return -1;
                        }

                        printf("get afer jerasure_make_decoding_matrix=========================================\n");
                        
                        CUDA_CHECK_RETURN( hipMemcpy(decode_matrix_gpu, decode_matrix, k*k*sizeof(int), hipMemcpyHostToDevice) );
                        CUDA_CHECK_RETURN( hipMemcpy(dm_ids_gpu, dm_ids, k*sizeof(int), hipMemcpyHostToDevice) );
                        

                        erased_data_size = full_erased_list_data( k, m, erasure_loc_data, erased );
                        erased_coding_size = full_erased_list_coding( k, m, erasure_loc_coding, erased );

                        CUDA_CHECK_RETURN( hipMemcpy(erasure_loc_data_gpu, erasure_loc_data, qt*sizeof(int), hipMemcpyHostToDevice) );
                        CUDA_CHECK_RETURN( hipMemcpy(erasure_loc_coding_gpu, erasure_loc_coding, qt*sizeof(int), hipMemcpyHostToDevice) );
                   
                    }

                    //dout(10) << "going to decode for B's in repair plane "<< *z << " at index " << repair_plane_to_ind[*z] << dendl;
                    //jerasure_matrix_decode_substripe(k+nu, m, w, matrix, 0, erasure_locations, &B_buf[0], &B_buf[k+nu], repair_plane_to_ind[*z], sub_chunksize);
                                            
                    planeKernel<<<planeKernelGridSize,planeKernelBlockSize,(k*k + k * m)*sizeof(char),streams[1]>>>(gf_table, \
                        k, clmsrProfileP->nu, m, clmsrProfileP->w, q, t,\
                        clmsrGpuP->matrix_gpu, decode_matrix_gpu, dm_ids_gpu, erasure_loc_data_gpu, erased_data_size, erasure_loc_coding_gpu, erased_coding_size, planePOnGpuK, pieceSize, pieceSizeMax\
                     );//decode;


                    //debug
                    CUDA_CHECK_RETURN( hipDeviceSynchronize() );

                    for(int i = 0; i < num_erased; i++)
                    {
                        x = erasure_locations[i]%q;
                        y = erasure_locations[i]/q;
                        //dout(10) << "B symbol recovered at (x,y) = (" << x <<","<<y<<")"<<dendl;
                        //dout(10) << "erasure location " << erasure_locations[i] << dendl;
                        node_sw = y*q+z_vec[y];
                        z_sw = (*z) + (x - z_vec[y]) * (int)pow(q,t-1-y);


                        

                        //make sure it is not an aloof node before you retrieve repaired_data
                        if( aloof_nodes.find(erasure_locations[i]) == aloof_nodes.end())
                        {
                            if(x == z_vec[y] )
                            {//hole-dot pair (type 0)
                            //dout(10) << "recovering the hole dot pair/lost node in repair plane" << dendl;
                                A1 = &repaired_data[erasure_locations[i]][*z*sub_chunksize + pieceOffset];
                                CUDA_CHECK_RETURN( hipMemcpy( A1, planeOnGpu[erasure_locations[i]] + pieceSizeMax*2, pieceSize, hipMemcpyDeviceToHost) );
                                //memcpy(A1, B1, sub_chunksize);
                                count_retrieved_sub_chunks++;
                            }//can recover next case (type 2) only after obtaining B's for all the planes with same order
                            else
                            {
                                //恢复的是个want_to_read节点
                                if(repaired_data.find(erasure_locations[i]) != repaired_data.end() )
                                {//this is a hole (lost node)
                                    A1 = &repaired_data[erasure_locations[i]][*z*sub_chunksize + pieceOffset];
                                    //check if type-2
                                    //node_sw也是一个want_to_read节点
                                    if( repaired_data.find(node_sw) != repaired_data.end())
                                    {
                                        if(x < z_vec[y])//todo: check this is ensure!!!!
                                        {//recover both A1 and A2 here
                                            A2 = &repaired_data[node_sw][z_sw*sub_chunksize + pieceOffset];
                                            //B2 = &B_buf[node_sw][repair_plane_to_ind[z_sw]*sub_chunksize + pieceOffset];
                                            

                                            pieceKernelGamma<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,erasure_locations[i],GAMMA_INVERSE, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                            
                                            //debug
                                            CUDA_CHECK_RETURN( hipDeviceSynchronize() );

                                            CUDA_CHECK_RETURN( hipMemcpy( A1, planeOnGpu[erasure_locations[i]],                  pieceSize, hipMemcpyDeviceToHost) );
                                            CUDA_CHECK_RETURN( hipMemcpy( A2, planeOnGpu[erasure_locations[i]] + pieceSizeMax*1, pieceSize, hipMemcpyDeviceToHost) );
                                
                                            
                                            //gamma_inverse_transform(A1, A2, B1, B2, sub_chunksize);
                                            count_retrieved_sub_chunks = count_retrieved_sub_chunks + 2;
                                        }
                                        else
                                        {
                                            B1 = &B_buf[erasure_locations[i]][repair_plane_to_ind[*z]*sub_chunksize + pieceOffset];
                                            CUDA_CHECK_RETURN( hipMemcpy( B1, planeOnGpu[erasure_locations[i]] + pieceSizeMax*2, pieceSize, hipMemcpyDeviceToHost) );
                                        }
                                    }
                                    else//node_sw是一个helper节点：　这里不可能是一个aloof节点，因为同一个y-section的都是helper节点．
                                    {
                                        //dout(10) << "repaired_data" << repaired_data << dendl;
                                        //A2 for this particular node is available
                                        assert(helper_data.find(node_sw) != helper_data.end());
                                        assert(repair_plane_to_ind.find(z_sw) !=  repair_plane_to_ind.end());
                                        //A2 = &helper_data[node_sw][repair_plane_to_ind[z_sw]*sub_chunksize + pieceOffset];

                                        pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,erasure_locations[i],B1A2_A1, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                        
                                        //debug
                                        CUDA_CHECK_RETURN( hipDeviceSynchronize() );
                                        
                                        CUDA_CHECK_RETURN( hipMemcpy( A1, planeOnGpu[erasure_locations[i]], pieceSize, hipMemcpyDeviceToHost) );
                            
                                        //get_type1_A(A1, B1, A2, sub_chunksize);
                                        count_retrieved_sub_chunks++;
                                    }
                                }
                                else
                                {
                                    //这里是说我是一个helper,但我的z_sw是一个want_to_read,所以我需要恢复A2;
                                    //not a hole and has an erasure in the y-crossection.
                                    assert(repaired_data.find(node_sw) != repaired_data.end());
                                    if(repair_plane_to_ind.find(z_sw) == repair_plane_to_ind.end())
                                    {
                                        A2 = &repaired_data[node_sw][z_sw*sub_chunksize + pieceOffset];
                                        //i got to recover A2, if z_sw was already there
                                        //dout(10) << "recovering A2 of node:" << node_sw << " at location " << z_sw << dendl;
                                        pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,erasure_locations[i],A1B1_A2, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                        
                    //debug
                    CUDA_CHECK_RETURN( hipDeviceSynchronize() );


                                        CUDA_CHECK_RETURN( hipMemcpy( A2, planeOnGpu[erasure_locations[i]] + pieceSizeMax*1, pieceSize, hipMemcpyDeviceToHost) );
                            

                                        //A1 = &helper_data[erasure_locations[i]][repair_plane_to_ind[*z]*sub_chunksize];

                                        //get_type2_A(A2, B1, A1, sub_chunksize);
                                        count_retrieved_sub_chunks++;
                                    }
                                }
                            }//type-1 erasure recovered.
                        }//not an aloof node
                    }//erasures

           //dout(10) << "repaired data after decoding at plane: " << *z << " "<< repaired_data << dendl;
           //dout(10) << "helper data after decoding at plane: " << *z << " "<< helper_data << dendl;
                    pieceOffset += pieceSize;
                }
            }//planes of a particular order

        }
    }
    assert(repair_sub_chunks_ind.size() == (unsigned)plane_count);
    assert(sub_chunk_no*repaired_data.size() == (unsigned)count_retrieved_sub_chunks);

    //dout(10) << "repaired_data = " << repaired_data << dendl;

    //todo : something route like this
    //todo : ensure that size is big enough.

    /*
    for layers
    {
        for node
        {
            for pieces
            {
                tranfer node;
                cal node'b
                add result to RS
                transfer necessary b back;
            }
        }
        cal the A of origin Miss part
        transfer it back
    }
    */
    return 0;
}


int is_erasure_type_1(int m, int ind, erasure_t_gpu* erasures, int* z_vec){

  // Need to look for the column of where erasures[i] is and search to see if there is a hole dot pair.
  int i;

  if(erasures[ind].x == z_vec[erasures[ind].y]) return 0; //type-0 erasure

  for(i=0; i < m; i++){
    if(erasures[i].y == erasures[ind].y){
      if(erasures[i].x == z_vec[erasures[i].y]){
    return 0;
      }
    }
  }
  return 1;

}

void get_erasure_coordinates_gpu(int m, int q, int t, int* erasure_locations, erasure_t_gpu* erasures )
{
  int i;

  for(i = 0; i<m; i++){
    if(erasure_locations[i]==-1)break;
    erasures[i].x = erasure_locations[i]%q;
    erasures[i].y = erasure_locations[i]/q;
  }
}

int SingleGpuRoute::doDecode \
( int* erasure_locations, char** data_ptrs, char** code_ptrs, int* erased, \
                            int num_erasures, int* order, int* weight_vec, int max_weight, int size, char ** B_buf)

{
    FT(SingleGpuRoute::doDecode);
    init_gf_log_w8_gpu();

    erasure_t_gpu erasures[clmsrProfileP->m];
    get_erasure_coordinates_gpu(clmsrProfileP->m, clmsrProfileP->q, clmsrProfileP->t, erasure_locations, erasures);

    int i;

    char *A1 = NULL, *A2 = NULL;

    int hm_w;

    assert(size%clmsrProfileP->subChunkSize == 0);
    int ss_size = clmsrProfileP->subChunkSize;


   printf(" k: %d\t nu: %d\t m: %d\n",clmsrProfileP->k, clmsrProfileP->nu, clmsrProfileP->m );
    const int k = clmsrProfileP->k + clmsrProfileP->nu;
    const int m = clmsrProfileP->m;

    const int sub_chunksize = clmsrProfileP->subChunkSize;

    const int q = clmsrProfileP->q, t = clmsrProfileP->t, nu = clmsrProfileP->nu;
    const int qt = q * t;
    int* z_vec;

    CUDA_CHECK_RETURN( hipHostAlloc(&z_vec, t * sizeof(int), hipHostMallocPortable) );


    map<int, set<int> > ordered_planes;
    map<int, int> repair_plane_to_ind;
    int z, x,y, node_xy, node_sw, z_sw;
    int *decode_matrix_gpu;
    int *decode_matrix;
    int *dm_ids;
    int *dm_ids_gpu;
    int *erasure_loc_data;
    int *erasure_loc_data_gpu;
    int *erasure_loc_coding;
    int *erasure_loc_coding_gpu;
    int erased_data_size = 0;
    int erased_coding_size = 0;

    dm_ids = talloc(int, k);
    decode_matrix = talloc(int, k*k);
    erasure_loc_data = talloc(int, qt);
    erasure_loc_coding = talloc(int, qt);

    CUDA_CHECK_RETURN(hipMalloc(&decode_matrix_gpu, k * k *sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&dm_ids_gpu, k * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&erasure_loc_data_gpu, qt * sizeof(int)));
    CUDA_CHECK_RETURN(hipMalloc(&erasure_loc_coding_gpu, qt * sizeof(int)));

    //malloc the data temp space on the gpu a piece has 4 things: A1,A2,B1,B2;
    
    int pieceSizeMax = __getPieceSize(0);
    unsigned char* planeOnGpu[qt];
    for( int i = 0; i < qt; i ++ )
    {
        CUDA_CHECK_RETURN(hipMalloc(&planeOnGpu[i], pieceSizeMax*4));
    }

    unsigned char** planePOnGpuK;

    CUDA_CHECK_RETURN(hipMalloc(&planePOnGpuK, qt*sizeof(unsigned char*)));

    CUDA_CHECK_RETURN(hipMemcpy(planePOnGpuK, planeOnGpu, qt*sizeof(unsigned char*), hipMemcpyHostToDevice));

    //init decode vars: etc: decode_matrix
    printf("get before jerasure_make_decoding_matrix=========================================\n");
    if (jerasure_make_decoding_matrix(k, m, clmsrProfileP->w, clmsrProfileP->matrix, erased, decode_matrix, dm_ids) < 0) 
    {
      printf("Can not get decoding matrix!!!\n");
      return -1;
    }

    CUDA_CHECK_RETURN( hipMemcpy(decode_matrix_gpu, decode_matrix, k*k*sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK_RETURN( hipMemcpy(dm_ids_gpu, dm_ids, k*sizeof(int), hipMemcpyHostToDevice) );
    

    erased_data_size = full_erased_list_data( k, m, erasure_loc_data, erased );
    erased_coding_size = full_erased_list_coding( k, m, erasure_loc_coding, erased );

    CUDA_CHECK_RETURN( hipMemcpy(erasure_loc_data_gpu, erasure_loc_data, qt*sizeof(int), hipMemcpyHostToDevice) );
    CUDA_CHECK_RETURN( hipMemcpy(erasure_loc_coding_gpu, erasure_loc_coding, qt*sizeof(int), hipMemcpyHostToDevice) );
    printf("get afer jerasure_make_decoding_matrix=========================================\n");


    for(hm_w = 0; hm_w <= max_weight; hm_w++)
    {
            int pieceOffset = 0;
            int pieceSize = 0;
            //init_matrix = false;

                //printf("get set<int>::iterator z=ordered_planes[order].begin(); z != ordered_planes[order].end()=========================================\n");

                printf("pieceCount %d \n", pieceCount );

        for( int pi = 0; pi < pieceCount; pi ++ )
        {
            pieceSize = __getPieceSize(pi);

            for(z = 0; z< clmsrProfileP->sub_chunk_no; z++)
            {
                if(order[z]==hm_w)
                {
                    get_plane_vector(q, t, z,z_vec);
                    //__decode_erasures(erasure_locations, z, z_vec, data_ptrs, code_ptrs, ss_size, B_buf, pi, pieceSize, pieceSizeMax, );
                    for(x=0; x < q; x++)
                    {
                        for(y=0; y<t; y++)
                        {
                            //todo: may not need because A1 is erasured error!!!
                            if( erased[y*q+x] == 1 )
                            {
                                node_xy = y*q+x; 
                                node_sw = y*q+z_vec[y];
                                z_sw = z + (x - z_vec[y]) * (int)pow(q,t-1-y);


                                
                                A1 = (node_xy < k+nu) ? &data_ptrs[node_xy][z*ss_size + pieceOffset] : &code_ptrs[node_xy-k-nu][z*ss_size + pieceOffset];
                                A2 = (node_sw < k+nu) ? &data_ptrs[node_sw][z_sw*ss_size + pieceOffset] : &code_ptrs[node_sw-k-nu][z_sw*ss_size + pieceOffset];

                                if(erased[node_xy] == 0)
                                { //if not an erasure 
                                    if(z_vec[y] != x)
                                    {//not a dot
                                        //get_B1_fromA1A2(&B_buf[node_xy][z*ss_size + pieceOffset], A1, A2, ss_size);
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy],                A1, pieceSize, hipMemcpyHostToDevice) );
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax, A2, pieceSize, hipMemcpyHostToDevice) );
                                        
                                        //todo: find a parameter
                                        pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma, planePOnGpuK, node_xy, A1A2_B1, pieceSize, pieceSizeMax, clmsrProfileP->w );

                                    }
                                    else
                                    { //dot
                                        CUDA_CHECK_RETURN( hipMemcpy(planeOnGpu[node_xy] + pieceSizeMax*2, A1, pieceSize, hipMemcpyHostToDevice) );
                                        //memcpy(&B_buf[node_xy][z*ss_size  + pieceOffset], A1, ss_size);
                                    }
                                }
                            }
                        }
                    }

                    //Decode in B's
                   /* jerasure_matrix_decode_substripe(k+nu, m, w, matrix, 0, erasure_locations, 
                                                   &B_buf[0], &B_buf[k+nu], z, ss_size);
                   */
                    planeKernel<<<planeKernelGridSize,planeKernelBlockSize,(k*k + k * m)*sizeof(char),streams[1]>>>(gf_table, \
                        k, clmsrProfileP->nu, m, clmsrProfileP->w, q, t,\
                        clmsrGpuP->matrix_gpu, decode_matrix_gpu, dm_ids_gpu, erasure_loc_data_gpu, erased_data_size, erasure_loc_coding_gpu, erased_coding_size, planePOnGpuK, pieceSize, pieceSizeMax\
                     );
                    //end

                }
            }

        /* Need to get A's from B's*/
            for(z = 0; z< clmsrProfileP->sub_chunk_no; z++)
            {
                if(order[z]==hm_w)
                {
                    get_plane_vector(q, t, z, z_vec);
                    for(i = 0; i<num_erasures; i++)
                    {
                        x = erasures[i].x;
                        y = erasures[i].y;
                        node_xy = y*q+x;
                        node_sw = y*q+z_vec[y];
                        z_sw = z + ( x - z_vec[y] ) * (int)pow(q,t-1-y);

                        A1 = (node_xy < k+nu) ? &data_ptrs[node_xy][z*ss_size + pieceOffset] : &code_ptrs[node_xy-k-nu][z*ss_size  + pieceOffset];
                        A2 = (node_sw < k+nu) ? &data_ptrs[node_sw][z_sw*ss_size + pieceOffset] : &code_ptrs[node_sw-k-nu][z_sw*ss_size + pieceOffset];

                        if(z_vec[y] != x)
                        { //not a hole-dot pair
                            if(is_erasure_type_1(m, i, erasures, z_vec))
                            {
                                pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,erasure_locations[i],B1A2_A1, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                        
                                        //debug
                                CUDA_CHECK_RETURN( hipDeviceSynchronize() );
                                        
                                CUDA_CHECK_RETURN( hipMemcpy( A1, planeOnGpu[erasure_locations[i]], pieceSize, hipMemcpyDeviceToHost) );
                                //get_type1_A(A1, &B_buf[node_xy][z*ss_size], A2, ss_size);
                            }
                            else
                            {
              // case for type-2 erasure, there is a hole-dot pair in this y column
                                assert(erased[node_sw]==1);
                                //pieceKernelGamma<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,erasure_locations[i],GAMMA_INVERSE, pieceSize, pieceSizeMax, clmsrProfileP->w);
                                pieceKernel<<<pieceKernelGridSize,pieceKernelBlockSize,0,streams[1]>>>( gf_table, clmsrProfileP->gamma,planePOnGpuK,erasure_locations[i],B1B2_A1, pieceSize, pieceSizeMax, clmsrProfileP->w);            
                                            //debug
                                CUDA_CHECK_RETURN( hipDeviceSynchronize() );

                                CUDA_CHECK_RETURN( hipMemcpy( A1, planeOnGpu[erasure_locations[i]], pieceSize, hipMemcpyDeviceToHost) );
                                //get_A1_fromB1B2(A1, &B_buf[node_xy][z*ss_size], &B_buf[node_sw][z_sw*ss_size], ss_size);
                            }
                        }
                        else
                        { //for type 0 erasure (hole-dot pair)  copy the B1 to A1
                            CUDA_CHECK_RETURN( hipMemcpy( A1, planeOnGpu[erasure_locations[i]] + pieceSizeMax*2, pieceSize, hipMemcpyDeviceToHost) );   
                            //memcpy(A1, &B_buf[node_xy][z*ss_size], ss_size);
                        }

                    }//get A's from B's
                }
            }//plane

            pieceOffset += pieceSize;
        }
    }//hm_w, order

    return 0;
}

void SingleGpuRoute::deinit()
{
    for (int i =0; i < STREAM_NUM; ++i)
    {
        CUDA_CHECK_RETURN(hipStreamDestroy(streams[i]));
    }

    for (int i =0; i < EVENT_NUM; ++i)
    {
        CUDA_CHECK_RETURN(hipEventDestroy(events[i]));
    }
}

int SingleGpuRoute::init_gf_log_w8_gpu( hipStream_t stream )
{
/*    int ret = copy_log_to_gpu_w8( gf_table );

    unsigned char log_temp[256];
    unsigned char anti_temp[256*2];
    unsigned char inv_temp[256];

    //debug
    CUDA_CHECK_RETURN( hipMemcpy(log_temp, gf_table.g_log, 256, hipMemcpyDeviceToHost) );
    CUDA_CHECK_RETURN( hipMemcpy(anti_temp, gf_table.g_anti_log, 256*2, hipMemcpyDeviceToHost) );
    CUDA_CHECK_RETURN( hipMemcpy(inv_temp, gf_table.g_inv, 256, hipMemcpyDeviceToHost) );

    //cout << " mypt:  " << (int)(&c_log[0]) << endl;
    //printf("your pt0:   %d\n", &c_log[0] );

/*    for( int i = 0; i < 256; i ++ )
    {
        printf("temp---i: %d:\t%d\t %d\n", i, log_temp[i], anti_temp[i] );
    }

    for( int i = 256; i < 256*2; i ++ )
    {
        printf("temp---i: %d:\t%d\n", i, anti_temp[i] );
    }*/


    return copy_log_to_gpu_w8( gf_table );
}

void SingleGpuRoute::compareGf()
{
    init_gf_log_w8_gpu();
    //testGf<<<1,10>>>(gf_table,8);
    testGfHost( 8 );
}
